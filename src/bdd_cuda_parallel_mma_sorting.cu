#include "hip/hip_runtime.h"
#include "bdd_cuda_parallel_mma_sorting.h"
#include <thrust/sort.h>
#include <thrust/for_each.h>
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>

namespace LPMP {
    void bdd_cuda_parallel_mma_sorting::iteration()
    {
        initialize_costs();
        forward_run();
        backward_run();
    }

    void bdd_cuda_parallel_mma_sorting::solve(const size_t max_iter, const double tolerance, const double time_limit)
    {
        const auto start_time = std::chrono::steady_clock::now();
        double lb_prev = this->lower_bound();
        double lb_post = lb_prev;
        std::cout << "initial lower bound = " << lb_prev;
        auto time = std::chrono::steady_clock::now();
        std::cout << ", time = " << (double) std::chrono::duration_cast<std::chrono::milliseconds>(time - start_time).count() / 1000 << " s";
        std::cout << "\n";
        for(size_t iter=0; iter<max_iter; ++iter)
        {
            iteration();
            lb_prev = lb_post;
            lb_post = this->lower_bound();
            std::cout << "iteration " << iter << ", lower bound = " << lb_post;
            time = std::chrono::steady_clock::now();
            double time_spent = (double) std::chrono::duration_cast<std::chrono::milliseconds>(time - start_time).count() / 1000;
            std::cout << ", time = " << time_spent << " s";
            std::cout << "\n";
            if (time_spent > time_limit)
            {
                std::cout << "Time limit reached." << std::endl;
                break;
            }
            if (std::abs(lb_prev-lb_post) < std::abs(tolerance*lb_prev))
            {
                std::cout << "Relative progress less than tolerance (" << tolerance << ")\n";
                break;
            }
        }
        std::cout << "final lower bound = " << this->lower_bound() << "\n"; 
    }
}