#include "hip/hip_runtime.h"
#include "bdd_cuda_parallel_mma.h"
#include "cuda_utils.h"
#include <chrono>

namespace LPMP {

    template<typename REAL>
    bdd_cuda_parallel_mma<REAL>::bdd_cuda_parallel_mma(const BDD::bdd_collection& bdd_col) : bdd_cuda_base<REAL>(bdd_col)
    {
        init();
    }

    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::init()
    {
        delta_lo_ = thrust::device_vector<REAL>(this->nr_variables());
        delta_hi_ = thrust::device_vector<REAL>(this->nr_variables());
        mm_lo_local_ = thrust::device_vector<REAL>(*std::max_element(this->cum_nr_layers_per_hop_dist_.begin(), this->cum_nr_layers_per_hop_dist_.end())); // size of largest layer.
        mm_diff_ = thrust::device_vector<REAL>(this->hi_cost_.size());
        // Copy from arc costs because it contains infinity for arcs to bot sink
        hi_cost_out_ = thrust::device_vector<REAL>(this->hi_cost_);
        lo_cost_out_ = thrust::device_vector<REAL>(this->lo_cost_);
    }

    template<typename REAL>
    struct compute_mm_diff_flush_mm_lo {
        REAL omega;
        __device__ void operator()(const thrust::tuple<REAL&, REAL&> t) const
        {
            REAL& mm_hi = thrust::get<0>(t);
            REAL& mm_lo = thrust::get<1>(t);
            mm_hi = omega * (mm_hi - mm_lo);
            mm_lo = HIP_INF_F;
        }
    };

    template<typename REAL>
    __global__ void min_marginals_from_directional_costs_cuda(const int cur_num_bdd_nodes, const int start_offset, const int start_offset_layer,
                                                            const int* const __restrict__ lo_bdd_node_index, 
                                                            const int* const __restrict__ hi_bdd_node_index, 
                                                            const int* const __restrict__ bdd_node_to_layer_map, 
                                                            const REAL* const __restrict__ lo_cost,
                                                            const REAL* const __restrict__ hi_cost,
                                                            const REAL* const __restrict__ cost_from_root,
                                                            const REAL* const __restrict__ cost_from_terminal,
                                                            REAL* __restrict__ mm_lo_local, REAL* __restrict__ mm_hi_local)
    {
        const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
        const int num_threads = blockDim.x * gridDim.x;
        for (int bdd_node_idx = start_index + start_offset; bdd_node_idx < cur_num_bdd_nodes + start_offset; bdd_node_idx += num_threads) 
        {
            const int next_lo_node = lo_bdd_node_index[bdd_node_idx];
            if (next_lo_node < 0) // will matter when one row contains multiple BDDs, otherwise the terminal nodes are at the end anyway.
                continue; // nothing needs to be done for terminal node.

            const int next_hi_node = hi_bdd_node_index[bdd_node_idx];

            const REAL cur_c_from_root = cost_from_root[bdd_node_idx];
            const int layer_idx = bdd_node_to_layer_map[bdd_node_idx];

            atomicMin(&mm_lo_local[layer_idx - start_offset_layer], cur_c_from_root + lo_cost[layer_idx] + cost_from_terminal[next_lo_node]);
            atomicMin(&mm_hi_local[layer_idx - start_offset_layer], cur_c_from_root + hi_cost[layer_idx] + cost_from_terminal[next_hi_node]);
        }
    }

    // This function does not need lo_path_costs and hi_path_costs to compute min-marginals.
    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::min_marginals_from_directional_costs(const int hop_index, const REAL omega, REAL* const mm_diff_ptr_with_start_offset)
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        
        const int num_nodes_processed = hop_index > 0 ? this->cum_nr_bdd_nodes_per_hop_dist_[hop_index - 1] : 0;
        const int end_node = this->cum_nr_bdd_nodes_per_hop_dist_[hop_index];
        const int cur_num_bdd_nodes = this->cum_nr_bdd_nodes_per_hop_dist_[hop_index] - num_nodes_processed;
        const int blockCount = ceil(cur_num_bdd_nodes / (REAL) NUM_THREADS);

        const int start_offset_layer = hop_index > 0 ? this->cum_nr_layers_per_hop_dist_[hop_index - 1]: 0;
        const int end_offset_layer = this->cum_nr_layers_per_hop_dist_[hop_index];
        const int cur_num_layers = end_offset_layer - start_offset_layer;

        min_marginals_from_directional_costs_cuda<<<blockCount, NUM_THREADS>>>(cur_num_bdd_nodes, num_nodes_processed, start_offset_layer,
                                                thrust::raw_pointer_cast(this->lo_bdd_node_index_.data()),
                                                thrust::raw_pointer_cast(this->hi_bdd_node_index_.data()),
                                                thrust::raw_pointer_cast(this->bdd_node_to_layer_map_.data()),
                                                thrust::raw_pointer_cast(this->lo_cost_.data()),
                                                thrust::raw_pointer_cast(this->hi_cost_.data()),
                                                thrust::raw_pointer_cast(this->cost_from_root_.data()),
                                                thrust::raw_pointer_cast(this->cost_from_terminal_.data()),
                                                thrust::raw_pointer_cast(mm_lo_local_.data()),
                                                mm_diff_ptr_with_start_offset);

        thrust::device_ptr<REAL> mm_lo_start(mm_lo_local_.data());
        thrust::device_ptr<REAL> mm_diff_start = thrust::device_pointer_cast(mm_diff_ptr_with_start_offset);

        auto first = thrust::make_zip_iterator(thrust::make_tuple(mm_diff_start, mm_lo_start));
        auto last = thrust::make_zip_iterator(thrust::make_tuple(mm_diff_start + cur_num_layers, mm_lo_start + cur_num_layers));

        thrust::for_each(first, last, compute_mm_diff_flush_mm_lo<REAL>({omega})); // Convert to min-marginal difference and set mm_lo_local_ to inf.

        #ifndef NDEBUG
            hipDeviceSynchronize();  // Not necessary, only to compute exact timing of this function.
        #endif
    }

    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::min_marginals_from_directional_costs(const int hop_index, const REAL omega)
    {
        const int start_offset_layer = hop_index > 0 ? this->cum_nr_layers_per_hop_dist_[hop_index - 1]: 0;
        min_marginals_from_directional_costs(hop_index, omega, thrust::raw_pointer_cast(mm_diff_.data() + start_offset_layer));
    }


    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::iteration()
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        forward_iteration(0.5);
        backward_iteration(0.5);
    }

    template<typename REAL>
    __global__ void forward_step_with_solve(const int cur_num_bdd_nodes, const int start_offset,
                                const int* const __restrict__ lo_bdd_node_index, 
                                const int* const __restrict__ hi_bdd_node_index, 
                                const int* const __restrict__ bdd_node_to_layer_map, 
                                const int* const __restrict__ primal_variable_index, 
                                const REAL* const __restrict__ delta_lo,
                                const REAL* const __restrict__ delta_hi,
                                const REAL* const __restrict__ mm_diff,
                                const REAL* const __restrict__ lo_cost_in,
                                const REAL* const __restrict__ hi_cost_in,
                                REAL* __restrict__ lo_cost_out,
                                REAL* __restrict__ hi_cost_out,
                                REAL* __restrict__ cost_from_root)
    {
        const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
        const int num_threads = blockDim.x * gridDim.x;
        for (int bdd_node_idx = start_index + start_offset; bdd_node_idx < cur_num_bdd_nodes + start_offset; bdd_node_idx += num_threads) 
        {
            const int next_lo_node = lo_bdd_node_index[bdd_node_idx];
            if (next_lo_node < 0) // will matter when one row contains multiple BDDs, otherwise the terminal nodes are at the end anyway.
                continue; // nothing needs to be done for terminal node.
            
            const int layer_idx = bdd_node_to_layer_map[bdd_node_idx];
            const REAL cur_mm_diff_hi_lo = mm_diff[layer_idx]; 
            const int cur_primal_idx = primal_variable_index[layer_idx];

            const REAL cur_lo_cost = lo_cost_in[layer_idx] + min(cur_mm_diff_hi_lo, 0.0f) + delta_lo[cur_primal_idx];
            const REAL cur_c_from_root = cost_from_root[bdd_node_idx];

            atomicMin(&cost_from_root[next_lo_node], cur_c_from_root + cur_lo_cost);

            const REAL cur_hi_cost = hi_cost_in[layer_idx] + min(-cur_mm_diff_hi_lo, 0.0f) + delta_hi[cur_primal_idx];
            const int next_hi_node = hi_bdd_node_index[bdd_node_idx];
            atomicMin(&cost_from_root[next_hi_node], cur_c_from_root + cur_hi_cost);

            lo_cost_out[layer_idx] = cur_lo_cost;
            hi_cost_out[layer_idx] = cur_hi_cost;
        }
    }

    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::forward_iteration(const REAL omega)
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        if(!this->backward_state_valid_)
            this->backward_run(false); //For the first iteration need to have costs from terminal. 
        
        // Clear states.
        this->flush_costs_from_root();
        flush_mm();

        const int num_steps = this->cum_nr_bdd_nodes_per_hop_dist_.size() - 1;
        int num_nodes_processed = 0;
        for (int s = 0; s < num_steps; s++)
        {
            // 1. Compute min-marginals using costs from root, costs from terminal and hi_costs, lo_costs for current hop
            min_marginals_from_directional_costs(s, omega);

            const int cur_num_bdd_nodes = this->cum_nr_bdd_nodes_per_hop_dist_[s] - num_nodes_processed;
            const int blockCount = ceil(cur_num_bdd_nodes / (float) NUM_THREADS);

            // 2. Subtract from hi_costs, update costs from root.
            forward_step_with_solve<<<blockCount, NUM_THREADS>>>(cur_num_bdd_nodes, num_nodes_processed,
                                                                thrust::raw_pointer_cast(this->lo_bdd_node_index_.data()),
                                                                thrust::raw_pointer_cast(this->hi_bdd_node_index_.data()),
                                                                thrust::raw_pointer_cast(this->bdd_node_to_layer_map_.data()),
                                                                thrust::raw_pointer_cast(this->primal_variable_index_.data()),
                                                                thrust::raw_pointer_cast(delta_lo_.data()),
                                                                thrust::raw_pointer_cast(delta_hi_.data()),
                                                                thrust::raw_pointer_cast(mm_diff_.data()),
                                                                thrust::raw_pointer_cast(this->lo_cost_.data()),
                                                                thrust::raw_pointer_cast(this->hi_cost_.data()),
                                                                thrust::raw_pointer_cast(this->lo_cost_out_.data()),
                                                                thrust::raw_pointer_cast(this->hi_cost_out_.data()),
                                                                thrust::raw_pointer_cast(this->cost_from_root_.data()));
            num_nodes_processed += cur_num_bdd_nodes;
        }
        thrust::swap(this->lo_cost_, lo_cost_out_);
        thrust::swap(this->hi_cost_, hi_cost_out_);
        compute_delta();
        normalize_delta();

        this->forward_state_valid_ = true;
        this->flush_backward_states();

        #ifndef NDEBUG
            hipDeviceSynchronize();  // Not necessary, only to compute exact timing of this function.
        #endif
    }

    template<typename REAL>
    __global__ void backward_step_with_solve(const int cur_num_bdd_nodes, const int start_offset, 
                                            const int* const __restrict__ lo_bdd_node_index, 
                                            const int* const __restrict__ hi_bdd_node_index, 
                                            const int* const __restrict__ bdd_node_to_layer_map, 
                                            const int* const __restrict__ primal_variable_index, 
                                            const REAL* const __restrict__ delta_lo,
                                            const REAL* const __restrict__ delta_hi,
                                            const REAL* const __restrict__ mm_diff,
                                            const REAL* const __restrict__ lo_cost_in,
                                            const REAL* const __restrict__ hi_cost_in,
                                            REAL* __restrict__ lo_cost_out,
                                            REAL* __restrict__ hi_cost_out,
                                            REAL* __restrict__ cost_from_terminal)
    {
        const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
        const int num_threads = blockDim.x * gridDim.x;
        for (int bdd_node_idx = start_index + start_offset; bdd_node_idx < cur_num_bdd_nodes + start_offset; bdd_node_idx += num_threads) 
        {
            const int next_lo_node = lo_bdd_node_index[bdd_node_idx];
            if (next_lo_node < 0)
                continue; // nothing needs to be done for terminal node.
            
            const int layer_idx = bdd_node_to_layer_map[bdd_node_idx];
            const REAL cur_mm_diff_hi_lo = mm_diff[layer_idx]; 
            const int cur_primal_idx = primal_variable_index[layer_idx];

            const REAL cur_hi_cost = hi_cost_in[layer_idx] + (min(-cur_mm_diff_hi_lo, 0.0f)) + (delta_hi[cur_primal_idx]);
            const REAL cur_lo_cost = lo_cost_in[layer_idx] + (min(cur_mm_diff_hi_lo, 0.0f)) + (delta_lo[cur_primal_idx]);

            const int next_hi_node = hi_bdd_node_index[bdd_node_idx];

            // Update costs from terminal:
            cost_from_terminal[bdd_node_idx] = min(cur_hi_cost + cost_from_terminal[next_hi_node], cur_lo_cost + cost_from_terminal[next_lo_node]);

            lo_cost_out[layer_idx] = cur_lo_cost;
            hi_cost_out[layer_idx] = cur_hi_cost;
        }
    }

    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::backward_iteration(const REAL omega)
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        assert(this->forward_state_valid_); 

        flush_mm();

        for (int s = this->cum_nr_bdd_nodes_per_hop_dist_.size() - 2; s >= 0; s--)
        {
            // 1. Compute min-marginals using costs from root, costs from terminal and hi_costs, lo_costs for current hop
            min_marginals_from_directional_costs(s, omega);

            const int start_offset = s > 0 ? this->cum_nr_bdd_nodes_per_hop_dist_[s - 1] : 0;

            const int cur_num_bdd_nodes = this->cum_nr_bdd_nodes_per_hop_dist_[s] - start_offset;
            const int blockCount = ceil(cur_num_bdd_nodes / (REAL) NUM_THREADS);

            // 2. Subtract from hi_costs, update costs from terminal.
            backward_step_with_solve<<<blockCount, NUM_THREADS>>>(cur_num_bdd_nodes, start_offset,
                                                                thrust::raw_pointer_cast(this->lo_bdd_node_index_.data()),
                                                                thrust::raw_pointer_cast(this->hi_bdd_node_index_.data()),
                                                                thrust::raw_pointer_cast(this->bdd_node_to_layer_map_.data()),
                                                                thrust::raw_pointer_cast(this->primal_variable_index_.data()),
                                                                thrust::raw_pointer_cast(delta_lo_.data()),
                                                                thrust::raw_pointer_cast(delta_hi_.data()),
                                                                thrust::raw_pointer_cast(mm_diff_.data()),
                                                                thrust::raw_pointer_cast(this->lo_cost_.data()),
                                                                thrust::raw_pointer_cast(this->hi_cost_.data()),
                                                                thrust::raw_pointer_cast(lo_cost_out_.data()),
                                                                thrust::raw_pointer_cast(hi_cost_out_.data()),
                                                                thrust::raw_pointer_cast(this->cost_from_terminal_.data()));
        }
        thrust::swap(this->lo_cost_, lo_cost_out_);
        thrust::swap(this->hi_cost_, hi_cost_out_);
        compute_delta();
        normalize_delta();

        this->flush_forward_states();
        this->backward_state_valid_ = true;

        #ifndef NDEBUG
            hipDeviceSynchronize();  // Not necessary, only to compute exact timing of this function.
        #endif
    }

    template<typename REAL>
    struct distribute_delta_func {
        const REAL* delta_lo;
        const REAL* delta_hi;
        __host__ __device__ void operator()(const thrust::tuple<int, REAL&, REAL&> t) const
        {
            const int primal_index = thrust::get<0>(t);
            if (primal_index == INT_MAX)
                return; // terminal node.

            REAL& lo_cost = thrust::get<1>(t);
            REAL& hi_cost = thrust::get<2>(t);
            lo_cost += delta_lo[primal_index];
            hi_cost += delta_hi[primal_index];
        }
    };

    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::distribute_delta()
    {
        assert(this->primal_variable_index_.size() == this->lo_cost_.size());
        assert(this->primal_variable_index_.size() == this->hi_cost_.size());
        assert(this->delta_lo_.size() == this->num_bdds_per_var_.size());
        assert(this->delta_hi_.size() == this->num_bdds_per_var_.size());
        assert(this->delta_hi_.size() == this->nr_vars_);

        auto first = thrust::make_zip_iterator(thrust::make_tuple(this->primal_variable_index_.begin(), this->lo_cost_.begin(), this->hi_cost_.begin()));
        auto last = thrust::make_zip_iterator(thrust::make_tuple(this->primal_variable_index_.end(), this->lo_cost_.end(), this->hi_cost_.end()));

        distribute_delta_func<REAL> func({thrust::raw_pointer_cast(delta_lo_.data()), thrust::raw_pointer_cast(delta_hi_.data())});

        thrust::for_each(first, last, func);
        this->flush_forward_states();
        this->flush_backward_states();

        thrust::fill(delta_lo_.begin(), delta_lo_.end(), 0.0f);
        thrust::fill(delta_hi_.begin(), delta_hi_.end(), 0.0f);
    }

    template<typename REAL> struct pos_part
    {
        __host__ __device__ REAL operator()(const REAL x) { return max(x, (REAL) 0); }
    };

    template<typename REAL> struct abs_neg_part
    {
        __host__ __device__ REAL operator()(const REAL x) { return -min(x, (REAL) 0); }
    };

    template<typename REAL>
    struct tuple_sum
    {
        __host__ __device__
        thrust::tuple<REAL, REAL> operator()(const thrust::tuple<REAL, REAL>& t0, const thrust::tuple<REAL, REAL>& t1)
        {
            return thrust::make_tuple(thrust::get<0>(t0) + thrust::get<0>(t1), thrust::get<1>(t0) + thrust::get<1>(t1));
        }
    };

    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::compute_delta()
    {
        auto first_val = thrust::make_zip_iterator(thrust::make_tuple(
            thrust::make_permutation_iterator(thrust::make_transform_iterator(mm_diff_.begin(), pos_part<REAL>()), this->primal_variable_sorting_order_.begin()),
            thrust::make_permutation_iterator(thrust::make_transform_iterator(mm_diff_.begin(), abs_neg_part<REAL>()), this->primal_variable_sorting_order_.begin())));

        auto first_out_val = thrust::make_zip_iterator(thrust::make_tuple(delta_hi_.begin(), delta_lo_.begin()));

        thrust::equal_to<int> binary_pred;
        auto new_end = thrust::reduce_by_key(this->primal_variable_index_sorted_.begin(), this->primal_variable_index_sorted_.end() - this->nr_bdds_, first_val, 
                            thrust::make_discard_iterator(), first_out_val, binary_pred, tuple_sum<REAL>());
        assert(thrust::distance(first_out_val, new_end.second) == delta_hi_.size());
        // thrust::reduce_by_key(thrust::make_permutation_iterator(this->primal_variable_index_.begin(), primal_variable_sorting_order_.begin()),
        //                     thrust::make_permutation_iterator(this->primal_variable_index_.end(), primal_variable_sorting_order_.end()), first_val, 
        //                     thrust::make_discard_iterator(), first_out_val, binary_pred, tuple_sum<REAL>()); // Uses less memory but slower.

    }

    template<typename REAL>
    struct normalize_delta_func {
        __host__ __device__ void operator()(const thrust::tuple<REAL&, REAL&, int> t) const
        {
            const int norm = thrust::get<2>(t);
            REAL& hi_cost = thrust::get<0>(t);
            hi_cost /= norm;
            REAL& lo_cost = thrust::get<1>(t);
            lo_cost /= norm;
        }
    };

    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::normalize_delta()
    {
        auto first = thrust::make_zip_iterator(thrust::make_tuple(this->delta_hi_.begin(), this->delta_lo_.begin(), this->num_bdds_per_var_.begin()));
        auto last = thrust::make_zip_iterator(thrust::make_tuple(this->delta_hi_.end(), this->delta_lo_.end(), this->num_bdds_per_var_.end()));
        thrust::for_each(first, last, normalize_delta_func<REAL>());
    }

    template<typename REAL>
    void bdd_cuda_parallel_mma<REAL>::flush_mm()
    {   // Makes min marginals INF so that they can be populated again by in-place minimization
        thrust::fill(mm_lo_local_.begin(), mm_lo_local_.end(), CUDART_INF_F_HOST);
        thrust::fill(mm_diff_.begin(), mm_diff_.end(), CUDART_INF_F_HOST);
    }

    template class bdd_cuda_parallel_mma<float>;
    template class bdd_cuda_parallel_mma<double>;
}
