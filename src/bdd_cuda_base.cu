#include "hip/hip_runtime.h"
#include "bdd_cuda_base.h"
#include "time_measure_util.h"
#include "cuda_utils.h"
#include <thrust/sort.h>
#include <thrust/for_each.h>
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>

namespace LPMP {

    struct assign_new_indices_func {
        const int* new_indices;
        __host__ __device__ void operator()(int& idx)
        {
            if(idx >= 0) // non-terminal nodes.
                idx = new_indices[idx];
        }
    };

    struct BDD_group {
        std::vector<size_t> bdd_indices_;
        size_t total_num_hops_ = 0;
        void insert(const size_t bdd_index, const int num_hops)
        {
            bdd_indices_.push_back(bdd_index);
            total_num_hops_ += num_hops;
        }
        bool operator<(const BDD_group& rhs) const { return total_num_hops_ > rhs.total_num_hops_; }
    };

    std::vector<int> group_bdds(const BDD::bdd_collection& bdd_col, const int max_num_groups)
    {
        MEASURE_FUNCTION_EXECUTION_TIME
        struct bdd_with_num_vars {
            size_t bdd_index_;
            size_t num_vars_in_bdd_;
            bool operator<(const bdd_with_num_vars& rhs) const { return (num_vars_in_bdd_ > rhs.num_vars_in_bdd_); }
        };
        std::vector<bdd_with_num_vars> bdds_to_schedule;
        bdds_to_schedule.reserve(bdd_col.nr_bdds());

        for(size_t bdd_idx=0; bdd_idx < bdd_col.nr_bdds(); ++bdd_idx)
            bdds_to_schedule.push_back(bdd_with_num_vars({bdd_idx, bdd_col.variables(bdd_idx).size() + 1})); // + 1 to account for terminal nodes.
        
        std::priority_queue<BDD_group> bdd_groups;
        std::sort(bdds_to_schedule.begin(), bdds_to_schedule.end()); // Puts largest BDD at start.
        for (const auto &largest_bdd: bdds_to_schedule)
        {
            if(bdd_groups.size() < max_num_groups)
            {
                BDD_group current_group; // create a new group with only one BDD.
                current_group.insert(largest_bdd.bdd_index_, largest_bdd.num_vars_in_bdd_);
                bdd_groups.push(current_group);
            }
            else
            {
                BDD_group current_group = bdd_groups.top(); // pop the smallest group and append current bdd into it.
                bdd_groups.pop();
                current_group.insert(largest_bdd.bdd_index_, largest_bdd.num_vars_in_bdd_);
                bdd_groups.push(current_group);
            }
        }
        // Now assign group index to each BDD:
        std::vector<int> bdd_group_indices(bdd_col.nr_bdds());
        int group_id = bdd_groups.size() - 1;
        while(!bdd_groups.empty())
        {
            const BDD_group current_group = bdd_groups.top(); // pop the smallest group.
            bdd_groups.pop();
            for (const size_t bdd_idx: current_group.bdd_indices_)
                bdd_group_indices[bdd_idx] = group_id;
            group_id--;
        }
        return bdd_group_indices;
    }

    template<typename REAL>
    bdd_cuda_base<REAL>::bdd_cuda_base(const BDD::bdd_collection& bdd_col)
    {
        MEASURE_FUNCTION_EXECUTION_TIME
        initialize(bdd_col);
        thrust::device_vector<int> bdd_node_hop_dist, bdd_node_group_idx;
        std::tie(bdd_node_hop_dist, bdd_node_group_idx) = populate_bdd_nodes(bdd_col, 1048576 * 4);
        reorder_bdd_nodes(bdd_node_hop_dist, bdd_node_group_idx);
        compress_bdd_nodes_to_layer(bdd_node_hop_dist);
        set_terminal_nodes_costs();
        print_num_bdd_nodes_per_hop();
    }

    template<typename REAL>
    void bdd_cuda_base<REAL>::initialize(const BDD::bdd_collection& bdd_col)
    {
        MEASURE_FUNCTION_EXECUTION_TIME
        nr_vars_ = [&]() {
            size_t max_v=0;
            for(size_t bdd_nr=0; bdd_nr<bdd_col.nr_bdds(); ++bdd_nr)
                max_v = std::max(max_v, bdd_col.min_max_variables(bdd_nr)[1]);
            return max_v+1;
        }();
        nr_bdds_ = bdd_col.nr_bdds();
        std::vector<int> primal_variable_counts(nr_vars_, 0);
        std::vector<int> num_vars_per_bdd;
        for(size_t bdd_idx=0; bdd_idx < bdd_col.nr_bdds(); ++bdd_idx)
        {
            const std::vector<size_t> cur_bdd_variables = bdd_col.variables(bdd_idx);
            for (const auto& var : cur_bdd_variables)
                primal_variable_counts[var]++;
            num_vars_per_bdd.push_back(cur_bdd_variables.size());
            num_dual_variables_ += cur_bdd_variables.size();
            nr_bdd_nodes_ += bdd_col.nr_bdd_nodes(bdd_idx);
        }
        num_bdds_per_var_ = thrust::device_vector<int>(primal_variable_counts.begin(), primal_variable_counts.end());
        num_vars_per_bdd_ = thrust::device_vector<int>(num_vars_per_bdd.begin(), num_vars_per_bdd.end());
        // Initialize data per BDD node: 
        hi_cost_ = thrust::device_vector<REAL>(nr_bdd_nodes_, 0.0);
        lo_cost_ = thrust::device_vector<REAL>(nr_bdd_nodes_, 0.0);
        cost_from_root_ = thrust::device_vector<REAL>(nr_bdd_nodes_);
        cost_from_terminal_ = thrust::device_vector<REAL>(nr_bdd_nodes_);
        hi_path_cost_ = thrust::device_vector<REAL>(nr_bdd_nodes_);
        lo_path_cost_ = thrust::device_vector<REAL>(nr_bdd_nodes_);
    }

    template<typename REAL>
    std::tuple<thrust::device_vector<int>, thrust::device_vector<int>> bdd_cuda_base<REAL>::populate_bdd_nodes(const BDD::bdd_collection& bdd_col, const int max_num_groups)
    {
        MEASURE_FUNCTION_EXECUTION_TIME
        const std::vector<int> bdd_group_indices = group_bdds(bdd_col, max_num_groups); // Compute for each BDD which group does it belong to.
        const int num_groups = *std::max_element(bdd_group_indices.begin(), bdd_group_indices.end()) + 1;
        assert(num_groups <= max_num_groups);
        assert(bdd_group_indices.size() == nr_bdds_);
        std::vector<int> cur_hop_dist_per_group(num_groups, 0);

        std::vector<int> primal_variable_index;
        primal_variable_index.reserve(nr_bdd_nodes_);
        std::vector<int> lo_bdd_node_index;
        lo_bdd_node_index.reserve(nr_bdd_nodes_);
        std::vector<int> hi_bdd_node_index;
        hi_bdd_node_index.reserve(nr_bdd_nodes_);
        std::vector<int> bdd_index;
        bdd_index.reserve(nr_bdd_nodes_);
        // Store hop distance from root node, so that all nodes with same hop distance can be processed in parallel:
        std::vector<int> bdd_node_hop_dist;
        bdd_node_hop_dist.reserve(nr_bdd_nodes_);
        std::vector<int> bdd_node_group_idx;
        bdd_node_group_idx.reserve(nr_bdd_nodes_);
        std::vector<int> root_nodes_indices;
        root_nodes_indices.reserve(nr_bdds_);
        std::vector<int> bot_sink_indices;
        bot_sink_indices.reserve(nr_bdds_);
        std::vector<int> top_sink_indices;
        top_sink_indices.reserve(nr_bdds_);

        int index_flat = 0;
        for(size_t bdd_idx=0; bdd_idx < bdd_col.nr_bdds(); ++bdd_idx)
        {
            assert(bdd_col.is_qbdd(bdd_idx));
            assert(bdd_col.is_reordered(bdd_idx));
            root_nodes_indices.push_back(index_flat);
            const int group_idx = bdd_group_indices[bdd_idx];
            // Initialize hop distance from current state of the group. Thus root nodes can still be at > 0 hop distance.
            int cur_hop_dist = cur_hop_dist_per_group[group_idx]; 
            const size_t storage_offset = bdd_col.offset(bdd_idx);
            size_t prev_var = bdd_col(bdd_idx, storage_offset).index;
            bool prev_terminal = false;
            for(size_t bdd_node_idx=0; bdd_node_idx < bdd_col.nr_bdd_nodes(bdd_idx); ++bdd_node_idx, ++index_flat)
            {
                const auto cur_instr = bdd_col(bdd_idx, bdd_node_idx + storage_offset);
                const size_t var = cur_instr.index;
                if(prev_var != var)
                {
                    assert(prev_var < var || cur_instr.is_terminal());
                    prev_var = var;
                    if(!prev_terminal || !cur_instr.is_terminal())
                        cur_hop_dist++; // both terminal nodes can have same hop distance.
                }
                if(!cur_instr.is_terminal())
                {
                    assert(bdd_node_idx < bdd_col.nr_bdd_nodes(bdd_idx) - 2); // only last two nodes can be terminal nodes. 
                    primal_variable_index.push_back(var);
                    lo_bdd_node_index.push_back(cur_instr.lo);
                    hi_bdd_node_index.push_back(cur_instr.hi);
                    prev_terminal = false;
                }
                else
                {
                    primal_variable_index.push_back(INT_MAX);
                    if (cur_instr.is_topsink())
                    {
                        top_sink_indices.push_back(index_flat);
                        lo_bdd_node_index.push_back(TOP_SINK_INDICATOR_CUDA);
                        hi_bdd_node_index.push_back(TOP_SINK_INDICATOR_CUDA);
                    }
                    else
                    {
                        bot_sink_indices.push_back(index_flat);
                        lo_bdd_node_index.push_back(BOT_SINK_INDICATOR_CUDA);
                        hi_bdd_node_index.push_back(BOT_SINK_INDICATOR_CUDA);
                    }
                    prev_terminal = true;
                    assert(bdd_node_idx >= bdd_col.nr_bdd_nodes(bdd_idx) - 2);
                }
                bdd_node_hop_dist.push_back(cur_hop_dist);
                bdd_index.push_back(bdd_idx);
                bdd_node_group_idx.push_back(group_idx);
            }
            cur_hop_dist_per_group[group_idx] = cur_hop_dist;
        }
        assert(root_nodes_indices.size() == nr_bdds_);
        assert(bot_sink_indices.size() == nr_bdds_);
        assert(top_sink_indices.size() == nr_bdds_);
        
        // copy to GPU
        primal_variable_index_ = thrust::device_vector<int>(primal_variable_index.begin(), primal_variable_index.end());
        bdd_index_ = thrust::device_vector<int>(bdd_index.begin(), bdd_index.end());
        lo_bdd_node_index_ = thrust::device_vector<int>(lo_bdd_node_index.begin(), lo_bdd_node_index.end());
        hi_bdd_node_index_ = thrust::device_vector<int>(hi_bdd_node_index.begin(), hi_bdd_node_index.end());
        root_indices_ = thrust::device_vector<int>(root_nodes_indices.begin(), root_nodes_indices.end());
        top_sink_indices_ = thrust::device_vector<int>(top_sink_indices.begin(), top_sink_indices.end());
        bot_sink_indices_ = thrust::device_vector<int>(bot_sink_indices.begin(), bot_sink_indices.end());
        thrust::device_vector<int> bdd_node_hop_dist_dev(bdd_node_hop_dist.begin(), bdd_node_hop_dist.end());
        thrust::device_vector<int> bdd_node_group_idx_dev(bdd_node_group_idx.begin(), bdd_node_group_idx.end());
        return {bdd_node_hop_dist_dev, bdd_node_group_idx_dev};
    }

    template<typename REAL>
    void bdd_cuda_base<REAL>::reorder_bdd_nodes(thrust::device_vector<int>& bdd_node_hop_dist_dev, thrust::device_vector<int>& bdd_node_group_idx_dev)
    {
        MEASURE_FUNCTION_EXECUTION_TIME
        // Make nodes with same hop distance, BDD depth and bdd index contiguous in that order.
        thrust::device_vector<int> sorting_order(nr_bdd_nodes_);
        thrust::sequence(sorting_order.begin(), sorting_order.end());
        
        // Sort the BDD nodes as per the following rules:
        // Nodes with less hop distance are always earlier. For equal hop distance:
        // -  Put all nodes with lower group index earlier (where lower group index corresponds to longer groups). For equal group index:
        // -- Put all nodes within same BDD together.

        auto first_key = thrust::make_zip_iterator(thrust::make_tuple(bdd_node_hop_dist_dev.begin(), bdd_node_group_idx_dev.begin(), bdd_index_.begin()));
        auto last_key = thrust::make_zip_iterator(thrust::make_tuple(bdd_node_hop_dist_dev.end(), bdd_node_group_idx_dev.begin(), bdd_index_.end()));

        auto first_bdd_val = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.begin(), sorting_order.begin(),
                                                                        lo_bdd_node_index_.begin(), hi_bdd_node_index_.begin()));
        thrust::sort_by_key(first_key, last_key, first_bdd_val);
        
        // Since the ordering is changed so all previously assigned indices need to be updated:
        thrust::device_vector<int> new_indices(sorting_order.size());
        thrust::scatter(thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(0) + sorting_order.size(), 
                        sorting_order.begin(), new_indices.begin());
        assign_new_indices_func func({thrust::raw_pointer_cast(new_indices.data())});
        thrust::for_each(lo_bdd_node_index_.begin(), lo_bdd_node_index_.end(), func);
        thrust::for_each(hi_bdd_node_index_.begin(), hi_bdd_node_index_.end(), func);
        thrust::for_each(root_indices_.begin(), root_indices_.end(), func);
        thrust::for_each(top_sink_indices_.begin(), top_sink_indices_.end(), func);
        thrust::for_each(bot_sink_indices_.begin(), bot_sink_indices_.end(), func);

        // Count number of BDD nodes per hop distance. Need for launching CUDA kernel with appropiate offset and threads:
        thrust::device_vector<int> dev_cum_nr_bdd_nodes_per_hop_dist(nr_bdd_nodes_);
        auto last_red = thrust::reduce_by_key(bdd_node_hop_dist_dev.begin(), bdd_node_hop_dist_dev.end(), thrust::make_constant_iterator<int>(1), 
                                                thrust::make_discard_iterator(), 
                                                dev_cum_nr_bdd_nodes_per_hop_dist.begin());
        dev_cum_nr_bdd_nodes_per_hop_dist.resize(thrust::distance(dev_cum_nr_bdd_nodes_per_hop_dist.begin(), last_red.second));

        // Convert to cumulative:
        thrust::inclusive_scan(dev_cum_nr_bdd_nodes_per_hop_dist.begin(), dev_cum_nr_bdd_nodes_per_hop_dist.end(), dev_cum_nr_bdd_nodes_per_hop_dist.begin());

        cum_nr_bdd_nodes_per_hop_dist_ = std::vector<int>(dev_cum_nr_bdd_nodes_per_hop_dist.size());
        thrust::copy(dev_cum_nr_bdd_nodes_per_hop_dist.begin(), dev_cum_nr_bdd_nodes_per_hop_dist.end(), cum_nr_bdd_nodes_per_hop_dist_.begin());
    }

    template<typename REAL>
    void bdd_cuda_base<REAL>::set_terminal_nodes_costs()
    {
        MEASURE_FUNCTION_EXECUTION_TIME
        // Set costs of top sinks to itself to 0:
        thrust::scatter(thrust::make_constant_iterator<float>(0.0), thrust::make_constant_iterator<float>(0.0) + top_sink_indices_.size(),
                        top_sink_indices_.begin(), cost_from_terminal_.begin());

        // Set costs of bot sinks to top to infinity:
        thrust::scatter(thrust::make_constant_iterator<float>(CUDART_INF_F_HOST), thrust::make_constant_iterator<float>(CUDART_INF_F_HOST) + bot_sink_indices_.size(),
                        bot_sink_indices_.begin(), cost_from_terminal_.begin());
    }

    // Removes redundant information in hi_costs, primal_index, bdd_index as it is duplicated across
    // multiple BDD nodes for each layer.
    template<typename REAL>
    void bdd_cuda_base<REAL>::compress_bdd_nodes_to_layer(const thrust::device_vector<int>& bdd_node_hop_dist_dev)
    {
        MEASURE_FUNCTION_EXECUTION_TIME
        thrust::device_vector<REAL> hi_cost_compressed(hi_cost_.size());
        thrust::device_vector<REAL> lo_cost_compressed(lo_cost_.size());
        thrust::device_vector<int> primal_index_compressed(primal_variable_index_.size()); 
        thrust::device_vector<int> bdd_index_compressed(bdd_index_.size());
        
        auto first_key = thrust::make_zip_iterator(thrust::make_tuple(bdd_node_hop_dist_dev.begin(), bdd_index_.begin(), primal_variable_index_.begin()));
        auto last_key = thrust::make_zip_iterator(thrust::make_tuple(bdd_node_hop_dist_dev.end(), bdd_index_.end(), primal_variable_index_.end()));

        auto first_out_key = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_discard_iterator(), bdd_index_compressed.begin(), primal_index_compressed.begin()));

        // Compute number of BDD nodes in each layer:
        bdd_layer_width_ = thrust::device_vector<int>(nr_bdd_nodes_);
        auto new_end = thrust::reduce_by_key(first_key, last_key, thrust::make_constant_iterator<int>(1), first_out_key, bdd_layer_width_.begin());
        const int out_size = thrust::distance(first_out_key, new_end.first);      

        // Assign bdd node to layer map:
        bdd_node_to_layer_map_ = thrust::device_vector<int>(out_size);
        thrust::sequence(bdd_node_to_layer_map_.begin(), bdd_node_to_layer_map_.end());
        bdd_node_to_layer_map_ = repeat_values(bdd_node_to_layer_map_, bdd_layer_width_);

        // Compress hi_costs_, lo_costs_ (although initially they are infinity, 0 resp.) and also populate how many BDD layers per hop dist.
        thrust::device_vector<int> bdd_hop_dist_compressed(out_size);
        auto first_cost_val = thrust::make_zip_iterator(thrust::make_tuple(hi_cost_.begin(), lo_cost_.begin(), bdd_node_hop_dist_dev.begin()));
        auto first_cost_val_compressed = thrust::make_zip_iterator(thrust::make_tuple(hi_cost_compressed.begin(), lo_cost_compressed.begin(), bdd_hop_dist_compressed.begin()));

        auto new_end_unique = thrust::unique_by_key_copy(first_key, last_key, first_cost_val, thrust::make_discard_iterator(), first_cost_val_compressed);
        assert(out_size == thrust::distance(first_cost_val_compressed, new_end_unique.second));

        hi_cost_compressed.resize(out_size);
        lo_cost_compressed.resize(out_size);
        primal_index_compressed.resize(out_size);
        bdd_index_compressed.resize(out_size);
        bdd_layer_width_.resize(out_size);

        thrust::swap(lo_cost_compressed, lo_cost_);
        thrust::swap(hi_cost_compressed, hi_cost_);
        thrust::swap(primal_index_compressed, primal_variable_index_);
        thrust::swap(bdd_index_compressed, bdd_index_);

        // For launching kernels where each thread operates on a BDD layer instead of a BDD node.
        layer_offsets_ = thrust::device_vector<int>(bdd_layer_width_.size() + 1);
        layer_offsets_[0] = 0;
        thrust::inclusive_scan(bdd_layer_width_.begin(), bdd_layer_width_.end(), layer_offsets_.begin() + 1);

        thrust::device_vector<int> dev_cum_nr_layers_per_hop_dist(cum_nr_bdd_nodes_per_hop_dist_.size());
        cum_nr_layers_per_hop_dist_ = std::vector<int>(dev_cum_nr_layers_per_hop_dist.size());

        thrust::reduce_by_key(bdd_hop_dist_compressed.begin(), bdd_hop_dist_compressed.end(), thrust::make_constant_iterator<int>(1), 
                            thrust::make_discard_iterator(), dev_cum_nr_layers_per_hop_dist.begin());

        thrust::inclusive_scan(dev_cum_nr_layers_per_hop_dist.begin(), dev_cum_nr_layers_per_hop_dist.end(), dev_cum_nr_layers_per_hop_dist.begin());
        thrust::copy(dev_cum_nr_layers_per_hop_dist.begin(), dev_cum_nr_layers_per_hop_dist.end(), cum_nr_layers_per_hop_dist_.begin());
    }

    template<typename REAL>
    void bdd_cuda_base<REAL>::flush_forward_states()
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        forward_state_valid_ = false;
        path_costs_valid_ = false;
    }

    template<typename REAL>
    void bdd_cuda_base<REAL>::flush_backward_states()
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        backward_state_valid_ = false;
        path_costs_valid_ = false;
    }

    template<typename REAL>
    void bdd_cuda_base<REAL>::print_num_bdd_nodes_per_hop()
    {
        int prev = 0;
        for(int i = 0; i < cum_nr_bdd_nodes_per_hop_dist_.size(); i++)
        {
            std::cout<<"Hop: "<<i<<", # BDD nodes: "<<cum_nr_bdd_nodes_per_hop_dist_[i] - prev<<std::endl;
            prev = cum_nr_bdd_nodes_per_hop_dist_[i];
        }
    }

    template<typename REAL>
    struct set_var_cost_func {
        int var_index;
        REAL cost;
        __device__ void operator()(const thrust::tuple<int, REAL&> t) const
        {
            const int cur_var_index = thrust::get<0>(t);
            if(cur_var_index != var_index)
                return;
            REAL& arc_cost = thrust::get<1>(t);
            arc_cost += cost;
        }
    };

    template<typename REAL>
    void bdd_cuda_base<REAL>::set_cost(const double c, const size_t var)
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        assert(var < nr_vars_);
        set_var_cost_func<REAL> func({(int) var, (REAL) c / num_bdds_per_var_[var]});

        auto first = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.begin(), hi_cost_.begin()));
        auto last = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.end(), hi_cost_.end()));

        thrust::for_each(first, last, func);
        flush_forward_states();
        flush_backward_states();
    }

    template<typename REAL>
    struct set_vars_costs_func {
        int* var_counts;
        REAL* primal_costs;
        __host__ __device__ void operator()(const thrust::tuple<int, REAL&> t) const
        {
            const int cur_var_index = thrust::get<0>(t);
            if (cur_var_index == INT_MAX)
                return; // terminal node.
            REAL& arc_cost = thrust::get<1>(t);
            const int count = var_counts[cur_var_index];
            assert(count > 0);
            arc_cost += primal_costs[cur_var_index] / count;
        }
    };

    template<typename REAL>
    template<typename COST_ITERATOR> 
    void bdd_cuda_base<REAL>::update_costs(COST_ITERATOR cost_lo_begin, COST_ITERATOR cost_lo_end, COST_ITERATOR cost_hi_begin, COST_ITERATOR cost_hi_end)
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
        assert(std::distance(cost_lo_begin, cost_lo_end) == nr_variables() || std::distance(cost_lo_begin, cost_lo_end) == 0);
        assert(std::distance(cost_hi_begin, cost_hi_end) == nr_variables() || std::distance(cost_hi_begin, cost_hi_end) == 0);

        auto populate_costs = [&](auto cost_begin, auto cost_end, auto base_cost_begin, auto base_cost_end) {
            thrust::device_vector<REAL> primal_costs(cost_begin, cost_end);

            set_vars_costs_func<REAL> func({thrust::raw_pointer_cast(num_bdds_per_var_.data()), 
                    thrust::raw_pointer_cast(primal_costs.data())});
            auto first = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.begin(), base_cost_begin));
            auto last = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.end(), base_cost_end));

            thrust::for_each(first, last, func);
        };

        if(std::distance(cost_lo_begin, cost_lo_end) > 0)
            populate_costs(cost_lo_begin, cost_lo_end, lo_cost_.begin(), lo_cost_.end());
        if(std::distance(cost_hi_begin, cost_hi_end) > 0)
            populate_costs(cost_hi_begin, cost_hi_end, hi_cost_.begin(), hi_cost_.end()); 

        flush_forward_states();
        flush_backward_states();
    }

    template void bdd_cuda_base<float>::update_costs(double*, double*, double*, double*);
    template void bdd_cuda_base<float>::update_costs(float*, float*, float*, float*);
    template void bdd_cuda_base<float>::update_costs(std::vector<double>::iterator, std::vector<double>::iterator, std::vector<double>::iterator, std::vector<double>::iterator);
    template void bdd_cuda_base<float>::update_costs(std::vector<double>::const_iterator, std::vector<double>::const_iterator, std::vector<double>::const_iterator, std::vector<double>::const_iterator);
    template void bdd_cuda_base<float>::update_costs(std::vector<float>::iterator, std::vector<float>::iterator, std::vector<float>::iterator, std::vector<float>::iterator);
    template void bdd_cuda_base<float>::update_costs(std::vector<float>::const_iterator, std::vector<float>::const_iterator, std::vector<float>::const_iterator, std::vector<float>::const_iterator);

    template void bdd_cuda_base<double>::update_costs(double*, double*, double*, double*);
    template void bdd_cuda_base<double>::update_costs(float*, float*, float*, float*);
    template void bdd_cuda_base<double>::update_costs(std::vector<double>::iterator, std::vector<double>::iterator, std::vector<double>::iterator, std::vector<double>::iterator);
    template void bdd_cuda_base<double>::update_costs(std::vector<double>::const_iterator, std::vector<double>::const_iterator, std::vector<double>::const_iterator, std::vector<double>::const_iterator);
    template void bdd_cuda_base<double>::update_costs(std::vector<float>::iterator, std::vector<float>::iterator, std::vector<float>::iterator, std::vector<float>::iterator);
    template void bdd_cuda_base<double>::update_costs(std::vector<float>::const_iterator, std::vector<float>::const_iterator, std::vector<float>::const_iterator, std::vector<float>::const_iterator);

    template<typename REAL>
    void bdd_cuda_base<REAL>::flush_costs_from_root()
    {
        thrust::fill(cost_from_root_.begin(), cost_from_root_.end(), CUDART_INF_F_HOST);
        // Set costs of root nodes to 0:
        thrust::scatter(thrust::make_constant_iterator<REAL>(0.0), thrust::make_constant_iterator<REAL>(0.0) + this->root_indices_.size(),
                        this->root_indices_.begin(), this->cost_from_root_.begin());
    }

    template<typename REAL>
    __global__ void forward_step(const int cur_num_bdd_nodes, const int start_offset,
                                const int* const __restrict__ lo_bdd_node_index, 
                                const int* const __restrict__ hi_bdd_node_index, 
                                const int* const __restrict__ bdd_node_to_layer_map, 
                                const REAL* const __restrict__ lo_cost,
                                const REAL* const __restrict__ hi_cost,
                                REAL* __restrict__ cost_from_root)
    {
        const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
        const int num_threads = blockDim.x * gridDim.x;
        for (int bdd_idx = start_index + start_offset; bdd_idx < cur_num_bdd_nodes + start_offset; bdd_idx += num_threads) 
        {
            const int next_lo_node = lo_bdd_node_index[bdd_idx];
            if (next_lo_node < 0)
                continue; // nothing needs to be done for terminal node.

            const int next_hi_node = hi_bdd_node_index[bdd_idx];

            const REAL cur_c_from_root = cost_from_root[bdd_idx];
            const int layer_idx = bdd_node_to_layer_map[bdd_idx];

            // Uncoalesced writes:
            atomicMin(&cost_from_root[next_lo_node], cur_c_from_root + lo_cost[layer_idx]);
            atomicMin(&cost_from_root[next_hi_node], cur_c_from_root + hi_cost[layer_idx]);
        }
    }

    template<typename REAL>
    void bdd_cuda_base<REAL>::forward_run()
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        if (forward_state_valid_)
            return;

        flush_costs_from_root();
        const int num_steps = cum_nr_bdd_nodes_per_hop_dist_.size() - 1;
        int num_nodes_processed = 0;
        for (int s = 0; s < num_steps; s++)
        {
            int threadCount = NUM_THREADS;
            int cur_num_bdd_nodes = cum_nr_bdd_nodes_per_hop_dist_[s] - num_nodes_processed;
            int blockCount = ceil(cur_num_bdd_nodes / (REAL) threadCount);
            forward_step<<<blockCount, threadCount>>>(cur_num_bdd_nodes, num_nodes_processed,
                                                    thrust::raw_pointer_cast(lo_bdd_node_index_.data()),
                                                    thrust::raw_pointer_cast(hi_bdd_node_index_.data()),
                                                    thrust::raw_pointer_cast(bdd_node_to_layer_map_.data()),
                                                    thrust::raw_pointer_cast(lo_cost_.data()),
                                                    thrust::raw_pointer_cast(hi_cost_.data()),
                                                    thrust::raw_pointer_cast(cost_from_root_.data()));
            num_nodes_processed += cur_num_bdd_nodes;
        }
        forward_state_valid_ = true;
    }

    template<typename REAL>
    __global__ void backward_step_with_path_costs(const int cur_num_bdd_nodes, const int start_offset,
                                                const int* const __restrict__ lo_bdd_node_index, 
                                                const int* const __restrict__ hi_bdd_node_index, 
                                                const int* const __restrict__ bdd_node_to_layer_map, 
                                                const REAL* const __restrict__ lo_cost,
                                                const REAL* const __restrict__ hi_cost,
                                                const REAL* __restrict__ cost_from_root, 
                                                REAL* __restrict__ cost_from_terminal,
                                                REAL* __restrict__ lo_path_cost, 
                                                REAL* __restrict__ hi_path_cost)
    {
        const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
        const int num_threads = blockDim.x * gridDim.x;
        for (int bdd_idx = start_index + start_offset; bdd_idx < cur_num_bdd_nodes + start_offset; bdd_idx += num_threads) 
        {
            const int lo_node = lo_bdd_node_index[bdd_idx];
            if (lo_node < 0)
                continue; // terminal node.
            const int hi_node = hi_bdd_node_index[bdd_idx];

            const int layer_idx = bdd_node_to_layer_map[bdd_idx];
            REAL cur_hi_cost_from_terminal = cost_from_terminal[hi_node] + hi_cost[layer_idx];
            REAL cur_lo_cost_from_terminal = cost_from_terminal[lo_node] + lo_cost[layer_idx];
            const REAL cur_cost_from_root = cost_from_root[bdd_idx];

            hi_path_cost[bdd_idx] = cur_cost_from_root + cur_hi_cost_from_terminal;
            lo_path_cost[bdd_idx] = cur_cost_from_root + cur_lo_cost_from_terminal;
            cost_from_terminal[bdd_idx] = min(cur_hi_cost_from_terminal, cur_lo_cost_from_terminal);
        }
    }

    template<typename REAL>
    __global__ void backward_step(const int cur_num_bdd_nodes, const int start_offset,
                                    const int* const __restrict__ lo_bdd_node_index, 
                                    const int* const __restrict__ hi_bdd_node_index, 
                                    const int* const __restrict__ bdd_node_to_layer_map, 
                                    const REAL* const __restrict__ lo_cost,
                                    const REAL* const __restrict__ hi_cost,
                                    REAL* __restrict__ cost_from_terminal)
    {
        const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
        const int num_threads = blockDim.x * gridDim.x;
        for (int bdd_idx = start_index + start_offset; bdd_idx < cur_num_bdd_nodes + start_offset; bdd_idx += num_threads) 
        {
            const int lo_node = lo_bdd_node_index[bdd_idx];
            if (lo_node < 0)
                continue; // terminal node.
            const int hi_node = hi_bdd_node_index[bdd_idx];

            const int layer_idx = bdd_node_to_layer_map[bdd_idx];
            cost_from_terminal[bdd_idx] = min(cost_from_terminal[hi_node] + hi_cost[layer_idx], cost_from_terminal[lo_node] + lo_cost[layer_idx]);
        }
    }


    template<typename REAL>
    void bdd_cuda_base<REAL>::backward_run(bool compute_path_costs)
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        if ((backward_state_valid_ && path_costs_valid_) ||
            (!compute_path_costs && backward_state_valid_))
            return;

        for (int s = cum_nr_bdd_nodes_per_hop_dist_.size() - 2; s >= 0; s--)
        {
            int threadCount = 256;
            int start_offset = 0;
            if(s > 0)
                start_offset = cum_nr_bdd_nodes_per_hop_dist_[s - 1];

            int cur_num_bdd_nodes = cum_nr_bdd_nodes_per_hop_dist_[s] - start_offset;
            int blockCount = ceil(cur_num_bdd_nodes / (REAL) threadCount);
            if (compute_path_costs)
                backward_step_with_path_costs<<<blockCount, threadCount>>>(cur_num_bdd_nodes, start_offset,
                                                        thrust::raw_pointer_cast(lo_bdd_node_index_.data()),
                                                        thrust::raw_pointer_cast(hi_bdd_node_index_.data()),
                                                        thrust::raw_pointer_cast(bdd_node_to_layer_map_.data()),
                                                        thrust::raw_pointer_cast(lo_cost_.data()),
                                                        thrust::raw_pointer_cast(hi_cost_.data()),
                                                        thrust::raw_pointer_cast(cost_from_root_.data()),
                                                        thrust::raw_pointer_cast(cost_from_terminal_.data()),
                                                        thrust::raw_pointer_cast(lo_path_cost_.data()),
                                                        thrust::raw_pointer_cast(hi_path_cost_.data()));
            else
                backward_step<<<blockCount, threadCount>>>(cur_num_bdd_nodes, start_offset,
                                                        thrust::raw_pointer_cast(lo_bdd_node_index_.data()),
                                                        thrust::raw_pointer_cast(hi_bdd_node_index_.data()),
                                                        thrust::raw_pointer_cast(bdd_node_to_layer_map_.data()),
                                                        thrust::raw_pointer_cast(lo_cost_.data()),
                                                        thrust::raw_pointer_cast(hi_cost_.data()),
                                                        thrust::raw_pointer_cast(cost_from_terminal_.data()));

        }
        backward_state_valid_ = true;
        if (compute_path_costs)
            path_costs_valid_ = true;
    }

    struct tuple_min
    {
        template<typename REAL>
        __host__ __device__
        thrust::tuple<REAL, REAL> operator()(const thrust::tuple<REAL, REAL>& t0, const thrust::tuple<REAL, REAL>& t1)
        {
            return thrust::make_tuple(min(thrust::get<0>(t0), thrust::get<0>(t1)), min(thrust::get<1>(t0), thrust::get<1>(t1)));
        }
    };

    // Computes min-marginals by reduction.
    template<typename REAL>
    std::tuple<thrust::device_vector<float>, thrust::device_vector<float>> bdd_cuda_base<REAL>::min_marginals_cuda()
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        forward_run();
        backward_run();

        auto first_val = thrust::make_zip_iterator(thrust::make_tuple(lo_path_cost_.begin(), hi_path_cost_.begin()));

        thrust::device_vector<float> min_marginals_lo(hi_cost_.size());
        thrust::device_vector<float> min_marginals_hi(hi_cost_.size());
        auto first_out_val = thrust::make_zip_iterator(thrust::make_tuple(min_marginals_lo.begin(), min_marginals_hi.begin()));

        thrust::equal_to<int> binary_pred;

        auto new_end = thrust::reduce_by_key(bdd_node_to_layer_map_.begin(), bdd_node_to_layer_map_.end(), first_val, thrust::make_discard_iterator(), first_out_val, binary_pred, tuple_min());
        const int out_size = thrust::distance(first_out_val, new_end.second);
        assert(out_size == hi_cost_.size());

        return {min_marginals_lo, min_marginals_hi};
    }

    template<typename REAL>
    two_dim_variable_array<std::array<double,2>> bdd_cuda_base<REAL>::min_marginals()
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        thrust::device_vector<float> mm_0, mm_1;

        std::tie(mm_0, mm_1) = min_marginals_cuda();

        // sort the min-marginals per bdd_index, primal_index:
        thrust::device_vector<int> bdd_index_sorted = bdd_index_;
        thrust::device_vector<int> primal_variable_index_sorted = primal_variable_index_;
        auto first_key = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_sorted.begin(), bdd_index_sorted.begin()));
        auto last_key = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_sorted.end(), bdd_index_sorted.end()));

        auto first_val = thrust::make_zip_iterator(thrust::make_tuple(mm_0.begin(), mm_1.begin()));

        thrust::sort_by_key(first_key, last_key, first_val);

        std::vector<int> num_bdds_per_var(num_bdds_per_var_.size());
        thrust::copy(num_bdds_per_var_.begin(), num_bdds_per_var_.end(), num_bdds_per_var.begin());

        std::vector<int> h_mm_primal_index(primal_variable_index_sorted.size());
        thrust::copy(primal_variable_index_sorted.begin(), primal_variable_index_sorted.end(), h_mm_primal_index.begin());

        std::vector<int> h_mm_bdd_index(bdd_index_sorted.size());
        thrust::copy(bdd_index_sorted.begin(), bdd_index_sorted.end(), h_mm_bdd_index.begin());

        std::vector<float> h_mm_0(mm_0.size());
        thrust::copy(mm_0.begin(), mm_0.end(), h_mm_0.begin());

        std::vector<float> h_mm_1(mm_1.size());
        thrust::copy(mm_1.begin(), mm_1.end(), h_mm_1.begin());

        std::vector<int> h_bdd_node_to_layer_map(bdd_node_to_layer_map_.size());
        thrust::copy(bdd_node_to_layer_map_.begin(), bdd_node_to_layer_map_.end(), h_bdd_node_to_layer_map.begin());

        two_dim_variable_array<std::array<double,2>> min_margs(num_bdds_per_var);

        for (int i = 0; i < nr_bdds_; ++i)
            assert(h_mm_primal_index[h_mm_primal_index.size() - 1 - i] == INT_MAX);
        int idx_1d = 0;
        for(int var = 0; var < nr_vars_; ++var)
        {
            assert(num_bdds_per_var[var] > 0);
            for(int bdd_idx = 0; bdd_idx < num_bdds_per_var[var]; ++bdd_idx, ++idx_1d)
            {
                assert(idx_1d < h_mm_primal_index.size() - nr_bdds_ && idx_1d < h_mm_0.size() - nr_bdds_ && idx_1d < h_mm_1.size() - nr_bdds_);
                assert(h_mm_primal_index[idx_1d] < INT_MAX); // Should ignore terminal nodes.
                std::array<double,2> mm = {h_mm_0[idx_1d], h_mm_1[idx_1d]};
                min_margs(var, bdd_idx) = mm;
            }
        }

        return min_margs;
    }

    template<typename REAL>
    void bdd_cuda_base<REAL>::update_costs(const thrust::device_vector<REAL>& update_vec)
    {
        thrust::transform(hi_cost_.begin(), hi_cost_.end(), update_vec.begin(), hi_cost_.begin(), thrust::plus<REAL>());
        flush_forward_states();
        flush_backward_states();
    }

    template<typename REAL>
    double bdd_cuda_base<REAL>::lower_bound()
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
        backward_run(false);
        // Sum costs_from_terminal of all root nodes:
        // fuse gather with reduction: 
        return thrust::reduce(thrust::make_permutation_iterator(cost_from_terminal_.begin(), root_indices_.begin()),
                                thrust::make_permutation_iterator(cost_from_terminal_.begin(), root_indices_.end()), 0.0);
    }

    template class bdd_cuda_base<float>;
    template class bdd_cuda_base<double>;

}
